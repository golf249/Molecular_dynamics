#include "hip/hip_runtime.h"
#include "../include_cuda/md.h"
#include <ctime>
#include <cmath>
#include <omp.h>
#include <iostream>

__global__ void computeLJForces(double* position, double* force_temp, int* type, int n) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n || i == j) return;

    double posx_i = position[3 * i];
    double posy_i = position[3 * i + 1];
    double posz_i = position[3 * i + 2];
    int type_i = type[i];

    double posx_j = position[3 * j];
    double posy_j = position[3 * j + 1];
    double posz_j = position[3 * j + 2];
    int type_j = type[j];

    double dx = posx_i - posx_j;
    double dy = posy_i - posy_j;
    double dz = posz_i - posz_j;
    double r2 = dx * dx + dy * dy + dz * dz;

    double sigma2, epsilon;
    if (type_i == 0 && type_j == 0) {
        sigma2 = 1.0;
        epsilon = 3.0;
    } else if (type_i == 1 && type_j == 1) {
        sigma2 = 9.0;
        epsilon = 60.0;
    } else {
        sigma2 = 4.0;
        epsilon = 15.0;
    }

    double sigma6 = sigma2 * sigma2 * sigma2;
    double sigma12 = sigma6 * sigma6;
    double inv_r2 = 1.0 / r2;
    double inv_r6 = inv_r2 * inv_r2 * inv_r2;
    double inv_r12 = inv_r6 * inv_r6;

    const double f = 24.0 * epsilon * inv_r2 * (2.0 * sigma12 * inv_r12 - sigma6 * inv_r6);

    force_temp[3 * (i * n + j)] = f * dx;
    force_temp[3 * (i * n + j) + 1] = f * dy;
    force_temp[3 * (i * n + j) + 2] = f * dz;
}

__global__ void sumForces(double* force_temp, double* force, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    double fx = 0.0, fy = 0.0, fz = 0.0;
    for (int j = 0; j < n; ++j) {
        fx += force_temp[3 * (i * n + j)];
        fy += force_temp[3 * (i * n + j) + 1];
        fz += force_temp[3 * (i * n + j) + 2];
    }

    force[3 * i] = fx;
    force[3 * i + 1] = fy;
    force[3 * i + 2] = fz;
}

// I acknowledge the use of ChatGPT 4.0 to generate an outline of how the structure of the MolecularDynamics class should be implemented.
// The majority of the content inside the methods was implemented by me. Where the code that was produced by AI is marked with a comment inside
// the relevant method, 

// Here AI was used to find the way to generate random number using the current time.
MolecularDynamics::MolecularDynamics(int numParticles, double dt, double Lx, double Ly, double Lz, int testCase, double temp, double percent_type1, double finalTime)
    : N(numParticles), dt(dt), Lx(Lx), Ly(Ly), Lz(Lz), testCase(testCase), temp(temp), percent_type1(percent_type1), finalTime(finalTime), 
    writeFile("particle_data.txt", "kinetic_energy.txt"), position_d(nullptr), force_d(nullptr), type_d(nullptr), d_force_temp(nullptr) {
    std::srand(static_cast<unsigned int>(std::time(nullptr))); // Seed the random number generator with the current time
    initialiseParticles();
}

MolecularDynamics::~MolecularDynamics() {
    // Free GPU memory
    hipFree(position_d);
    hipFree(force_d);
    hipFree(type_d);
    hipFree(d_force_temp);
}

/**
 * @brief Initialises the position, velocity and mass of particles for the molecular dynamics simulation based on the specified test case.
 * 
 * This function first clears any existing particles and initialises new particles based on the value of the `testCase` member variable.
 * 
 * Test cases:
 * - 1: Initialies a single particle at position (10.0, 10.0, 10.0) with zero velocity.
 * - 2: Initialises a single particle at position (10.0, 10.0, 10.0) with velocity (5.0, 2.0, 1.0).
 * - 3: Initialises two particles at positions (8.5, 10.0, 10.0) and (11.5, 10.0, 10.0) with zero velocity.
 * - 4: Initialises two particles at positions (8.5, 11.5, 10.0) and (11.5, 8.5, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively.
 * - 5: Initialises two particles at positions (8.5, 11.3, 10.0) and (11.5, 8.7, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively.
 * - 6: Initialises two particles at positions (8.5, 11.3, 10.0) and (11.5, 8.7, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively, and type 1 mass.
 * - Random: Initialises a specified number of particles of type 0 and type 1 with random positions and velocities, ensuring the positions between
 *  particles isn't <= 5 unit length. This is done via the `stabilityCheck` function.
 * 
 * The number of particles and its type is determined by the `percent_type1` and `N` member variables.
 * 
 * @note The random positions and velocities are generated using the `rand()` function and to ensure true randomness at every generation, a seed using the current time was used.
 */
// Here AI was used to generate the code to set the positions and velocity since 
// it is a repetitive task, but the structure of the conditional statement was implemented by me.
void MolecularDynamics::initialiseParticles() {
    particles.clear(); // Clear any existing particles

    if (testCase == 1) {
        particles = {
            Particle({10.0, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        std::cout << N << std::endl;
        // Allocate GPU memory on the device
        hipMallocManaged(&d_force_temp, 3 * N * N *sizeof(double));
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 2) {
        particles = {
            Particle({10.0, 10.0, 10.0}, {5.0, 2.0, 1.0}, 0)
        };
        int N = particles.size();
        std::cout << N << std::endl;
        // Allocate GPU memory on the device
        hipMallocManaged(&d_force_temp, 3 * N * N *sizeof(double));
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 3) {
        particles = {
            Particle({8.5, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0),
            Particle({11.5, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        std::cout << N << std::endl;
        // Allocate GPU memory on the device
        hipMallocManaged(&d_force_temp, 3 * N * N *sizeof(double));
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 4) {
        particles = {
            Particle({8.5, 11.5, 10.0}, {0.5, 0.0, 0.0}, 0),
            Particle({11.5, 8.5, 10.0}, {-0.5, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        std::cout << N << std::endl;
        // Allocate GPU memory on the device
        hipMallocManaged(&d_force_temp, 3 * N * N *sizeof(double));
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 5) {
        particles = {
            Particle({8.5, 11.3, 10.0}, {0.5, 0.0, 0.0}, 0),
            Particle({11.5, 8.7, 10.0}, {-0.5, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        std::cout << N << std::endl;
        // Allocate GPU memory on the device
        hipMallocManaged(&d_force_temp, 3 * N * N *sizeof(double));
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 6) {
        particles = {
            Particle({8.5, 11.3, 10.0}, {0.5, 0.0, 0.0}, 1),
            Particle({11.5, 8.7, 10.0}, {-0.5, 0.0, 0.0}, 1)
        };
        int N = particles.size();
        std::cout << N << std::endl;
        // Allocate GPU memory on the device
        hipMallocManaged(&d_force_temp, 3 * N * N *sizeof(double));
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else {
        int numType1 = static_cast<int>(std::ceil(percent_type1 / 100.0 * N));
        int numType0 = N - numType1;
        // Allocate GPU memory on the device
        hipMallocManaged(&d_force_temp, 3 * N * N *sizeof(double));
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
}
}


void MolecularDynamics::assignRandomStates(const int numType, const int type) {
    const double invRand = 1.0 / RAND_MAX;
    for (int i = 0; i < numType; ++i) {
        bool validPosition = false;
        std::array<double, 3> position, velocity;

        while (!validPosition) {
            position = { Lx * ((double)rand() * invRand), Ly * ((double)rand() * invRand), Lz * ((double)rand() * invRand) };
            velocity = { ((double)rand() * invRand - 0.5), ((double)rand() * invRand - 0.5), ((double)rand() * invRand - 0.5) };

            validPosition = stabilityCheck(position);
        }

        particles.emplace_back(position, velocity, type);
    }
}

// Here AI was used to outline implementation of how to detect whether particles are too close 
// to each other, specifically the norm of the difference between the positions of two particles.
bool MolecularDynamics::stabilityCheck(const std::array<double, 3>& position) {
    constexpr double R2 = 0.25; // Minimum allowed distance squared (0.5^2) {
    for (const Particle& p : particles) {
        const std::array<double, 3> otherPosition = p.getPosition();
        double dx = position[0] - otherPosition[0];
        double dy = position[1] - otherPosition[1];
        double dz = position[2] - otherPosition[2];
        if ((dx*dx + dy*dy + dz*dz) < R2) {
            return false;
        }
    }
    return true;
}

void MolecularDynamics::calForcesCUDA() {
    const int n = particles.size();
    std::cout << "yo" << std::endl;    

    // Populate the arrays in the allocated memory
    for (int i = 0; i < n; i++) {
        const std::array<double, 3>& pos = particles[i].getPosition();
        std::cout << "yo" << std::endl;    

        // Get the position of the particle
        position_d[3 * i] = pos[0];
        position_d[3 * i + 1] = pos[1];
        position_d[3 * i + 2] = pos[2];
        std::cout << "yo" << std::endl;    

        // Set the forces for each particle to 0
        force_d[3 * i] = 0.0;
        force_d[3 * i + 1] = 0.0;
        force_d[3 * i + 2] = 0.0;
        // Get the type of the particle
        type_d[i] = particles[i].getType();
    }
    std::cout << "yo" << std::endl;    

    int numThreads = 16;
    dim3 threadsPerBlock(numThreads, numThreads);
    dim3 numBlocks((n + numThreads - 1) / numThreads, (n + numThreads - 1) / numThreads);
    std::cout << "NumBlocks: " << numBlocks.x << " " << numBlocks.y << std::endl;
    // Compute LJ forces and store in temporary buffer
    computeLJForces<<<numBlocks, threadsPerBlock>>>(position_d, d_force_temp, type_d, n);
    hipDeviceSynchronize();  // Wait for kernel completion

    // Sum up forces into d_force
    dim3 sumBlocks((n + numThreads - 1) / numThreads);
    dim3 sumThreads(numThreads);
    sumForces<<<sumBlocks, sumThreads>>>(d_force_temp, force_d, n);
    hipDeviceSynchronize();  // Ensure summation is complete
}

void MolecularDynamics::forwardEuler() {
    for (Particle& p : particles) {
        std::array<double, 3> velocity = p.getVelocity();
        std::array<double, 3> position = p.getPosition();
        const std::array<double, 3> force = p.getForce();
        const double mass = p.getMass();
        for (int k = 0; k < 3; ++k) {
            velocity[k] += dt * force[k] / mass;
            position[k] += dt * velocity[k];
        }
        p.setVelocity(velocity);
        p.setPosition(position);
    }
    bcCheck();
}

void MolecularDynamics::bcCheck() {
    // Reflect particles if they go out of bounds
    for (Particle& p : particles) {
        std::array<double, 3> position = p.getPosition();
        std::array<double, 3> velocity = p.getVelocity();
        // For each coordinate: 0->Lx, 1->Ly, 2->Lz
        for (int k = 0; k < 3; ++k) {
            double L = (k == 0) ? Lx : ((k == 1) ? Ly : Lz); // Select the correct bound size for each axis
            if (position[k] < 0) {
                position[k] = -position[k];
                velocity[k] = std::abs(velocity[k]);
            } else if (position[k] > L) {
                position[k] = 2 * L - position[k];
                velocity[k] = -std::abs(velocity[k]);
            }
        }
        p.setPosition(position);
        p.setVelocity(velocity);
    }
}

/**
 * @brief Calculates the total kinetic energy of the system.
 *
 * This function iterates over all particles in the system at current 
 * time, retrieves their velocities, and computes the kinetic energy 
 * using the formula:
 * 
 *     KE = 0.5 * mass * (velocity_x^2 + velocity_y^2 + velocity_z^2)
 * 
 * The total kinetic energy is then accumulated and stored in the member
 * variable `kineticEnergy`.
 */
void MolecularDynamics::calKE() {
    kineticEnergy = 0.0;
    for (const Particle& p : particles) {
        const std::array<double, 3>& velocity = p.getVelocity();
        double speedSquared = velocity[0] * velocity[0] + velocity[1] * velocity[1] + velocity[2] * velocity[2]; 
        kineticEnergy += 0.5 * p.getMass() * speedSquared;
    }
}

void MolecularDynamics::velRescale() {
    // check if temp is not set else continue
    if (temp == -1.0) {
        return;
    }

    constexpr double kb = 0.8314459920816467; // Boltzmann constant
    double tempKE = (2.0 / (3.0 * kb * N)) * kineticEnergy;
    const double lambda = std::sqrt(temp / tempKE);

    for (Particle& p : particles) {
        std::array<double, 3> velocity = p.getVelocity();
        for (int k = 0; k < 3; ++k) {
            velocity[k] *= lambda;
        }
        p.setVelocity(velocity);
    }
}

void MolecularDynamics::runSimulation() {
    // Compute initial kinetic energy
    calKE();
    
    // Output initial conditions before starting integration such that
    // the initial state is also recorded at time 0.
    outputParticleData(0);
    outputKineticEnergy(0);

    const double outputTime = 0.1; // output time for the txt files
    const int steps = static_cast<int>(finalTime / dt); // simulation time steps
    const int outputStep = static_cast<int>(outputTime / dt); // file output time steps

    // Start simulation from step 1
    for (int step = 1; step <= steps; ++step) {
        const double currentTime = step * dt;
        calKE();

        if (testCase == -1) 
            velRescale();
        std::cout << "yo" << std::endl;    
        calForcesCUDA();
        // std::cout << "yo" << std::endl;    

        forwardEuler();

        // Output particle data and kinetic energy at 0.1 unit time intervals for the output files
        if (step % outputStep == 0) {
            if (testCase != -1) {
                outputParticleData(currentTime);
            }
            outputKineticEnergy(currentTime);
        }
    }
}

void MolecularDynamics::outputParticleData(double time) {
    for (size_t i = 0; i < particles.size(); ++i) {
        const Particle& p = particles[i];
        writeFile.writeParticleData(time, i, p.getPosition(), p.getVelocity());
    }
}

void MolecularDynamics::outputKineticEnergy(double time) {
    writeFile.writeKineticEnergy(time, kineticEnergy);
}
