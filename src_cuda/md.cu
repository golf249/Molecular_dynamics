#include "hip/hip_runtime.h"
#include "../include_cuda/md.h"
#include <ctime>
#include <cmath>
#include <omp.h>
#include <iostream>

// New kernel: each thread computes the net force on one particle.
__global__ void computeLJForces(double* position, double* force, int* type, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    
    double force_accum_x = 0.0, force_accum_y = 0.0, force_accum_z = 0.0;
    
    double posx_i = position[3 * i];
    double posy_i = position[3 * i + 1];
    double posz_i = position[3 * i + 2];
    int type_i = type[i];
    
    // Loop over all particles to compute interactions.
    for (int j = 0; j < n; ++j) {
        if (i == j) continue;
        
        double posx_j = position[3 * j];
        double posy_j = position[3 * j + 1];
        double posz_j = position[3 * j + 2];
        int type_j = type[j];
        
        double dx = posx_i - posx_j;
        double dy = posy_i - posy_j;
        double dz = posz_i - posz_j;
        double r2 = dx * dx + dy * dy + dz * dz;
        
        double sigma2, epsilon;
        if (type_i == 0 && type_j == 0) {
            sigma2 = 1.0;
            epsilon = 3.0;
        } else if (type_i == 1 && type_j == 1) {
            sigma2 = 9.0;
            epsilon = 60.0;
        } else {
            sigma2 = 4.0;
            epsilon = 15.0;
        }
        
        double sigma6 = sigma2 * sigma2 * sigma2;
        double sigma12 = sigma6 * sigma6;
        double inv_r2 = 1.0 / r2;
        double inv_r6 = inv_r2 * inv_r2 * inv_r2;
        double inv_r12 = inv_r6 * inv_r6;
        
        double f = 24.0 * epsilon * inv_r2 * (2.0 * sigma12 * inv_r12 - sigma6 * inv_r6);
        
        force_accum_x += f * dx;
        force_accum_y += f * dy;
        force_accum_z += f * dz;

        //printf("i,j %d,%d Force: (%f, %f, %f)\n", i, j, f * dx, dy, dz);
    }
    
    // Write the net force for particle i to global memory.
    force[3 * i]     = force_accum_x;
    force[3 * i + 1] = force_accum_y;
    force[3 * i + 2] = force_accum_z;
}

// I acknowledge the use of ChatGPT 4.0 to generate an outline of how the structure of the MolecularDynamics class should be implemented.
// The majority of the content inside the methods was implemented by me. Where the code that was produced by AI is marked with a comment inside
// the relevant method, 

// Here AI was used to find the way to generate random number using the current time.
MolecularDynamics::MolecularDynamics(int numParticles, double dt, double Lx, double Ly, double Lz, int testCase, 
    double temp, double percent_type1, double finalTime)
    : N(numParticles), dt(dt), Lx(Lx), Ly(Ly), Lz(Lz), testCase(testCase), temp(temp), percent_type1(percent_type1), finalTime(finalTime), 
    writeFile("particle_data.txt", "kinetic_energy.txt"), position_d(nullptr), force_d(nullptr), type_d(nullptr) {
    std::srand(static_cast<unsigned int>(std::time(nullptr))); 
    initialiseParticles();
}

MolecularDynamics::~MolecularDynamics() {
    // Free GPU memory
    hipFree(position_d);
    hipFree(force_d);
    hipFree(type_d);
}

/**
 * @brief Initialises the position, velocity and mass of particles for the molecular dynamics simulation based on the specified test case.
 * 
 * This function first clears any existing particles and initialises new particles based on the value of the `testCase` member variable.
 * 
 * Test cases:
 * - 1: Initialies a single particle at position (10.0, 10.0, 10.0) with zero velocity.
 * - 2: Initialises a single particle at position (10.0, 10.0, 10.0) with velocity (5.0, 2.0, 1.0).
 * - 3: Initialises two particles at positions (8.5, 10.0, 10.0) and (11.5, 10.0, 10.0) with zero velocity.
 * - 4: Initialises two particles at positions (8.5, 11.5, 10.0) and (11.5, 8.5, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively.
 * - 5: Initialises two particles at positions (8.5, 11.3, 10.0) and (11.5, 8.7, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively.
 * - 6: Initialises two particles at positions (8.5, 11.3, 10.0) and (11.5, 8.7, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively, and type 1 mass.
 * - Random: Initialises a specified number of particles of type 0 and type 1 with random positions and velocities, ensuring the positions between
 *  particles isn't <= 5 unit length. This is done via the `stabilityCheck` function.
 * 
 * The number of particles and its type is determined by the `percent_type1` and `N` member variables.
 * 
 * @note The random positions and velocities are generated using the `rand()` function and to ensure true randomness at every generation, a seed using the current time was used.
 */
// Here AI was used to generate the code to set the positions and velocity since 
// it is a repetitive task, but the structure of the conditional statement was implemented by me.
void MolecularDynamics::initialiseParticles() {
    particles.clear(); // Clear any existing particles

    if (testCase == 1) {
        particles = {
            Particle({10.0, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 2) {
        particles = {
            Particle({10.0, 10.0, 10.0}, {5.0, 2.0, 1.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 3) {
        particles = {
            Particle({8.5, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0),
            Particle({11.5, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 4) {
        particles = {
            Particle({8.5, 11.5, 10.0}, {0.5, 0.0, 0.0}, 0),
            Particle({11.5, 8.5, 10.0}, {-0.5, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 5) {
        particles = {
            Particle({8.5, 11.3, 10.0}, {0.5, 0.0, 0.0}, 0),
            Particle({11.5, 8.7, 10.0}, {-0.5, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 6) {
        particles = {
            Particle({8.5, 11.3, 10.0}, {0.5, 0.0, 0.0}, 1),
            Particle({11.5, 8.7, 10.0}, {-0.5, 0.0, 0.0}, 1)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else {
        int numType1 = static_cast<int>(std::ceil(percent_type1 / 100.0 * N));
        int numType0 = N - numType1;
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
}
}


void MolecularDynamics::assignRandomStates(const int numType, const int type) {
    const double invRand = 1.0 / RAND_MAX;
    for (int i = 0; i < numType; ++i) {
        bool validPosition = false;
        std::array<double, 3> position, velocity;

        while (!validPosition) {
            position = { Lx * ((double)rand() * invRand), Ly * ((double)rand() * invRand), Lz * ((double)rand() * invRand) };
            velocity = { ((double)rand() * invRand - 0.5), ((double)rand() * invRand - 0.5), ((double)rand() * invRand - 0.5) };

            validPosition = stabilityCheck(position);
        }

        particles.emplace_back(position, velocity, type);
    }
}

// Here AI was used to outline implementation of how to detect whether particles are too close 
// to each other, specifically the norm of the difference between the positions of two particles.
bool MolecularDynamics::stabilityCheck(const std::array<double, 3>& position) {
    constexpr double R2 = 0.25; // Minimum allowed distance squared (0.5^2) {
    for (const Particle& p : particles) {
        const std::array<double, 3> otherPosition = p.getPosition();
        double dx = position[0] - otherPosition[0];
        double dy = position[1] - otherPosition[1];
        double dz = position[2] - otherPosition[2];
        if ((dx*dx + dy*dy + dz*dz) < R2) {
            return false;
        }
    }
    return true;
}

void MolecularDynamics::calForcesCUDA() {
    const int n = particles.size();
    
    // Populate managed arrays from particle data.
    for (int i = 0; i < n; i++) {
        const std::array<double, 3>& pos = particles[i].getPosition();

        // Get the position of the particle
        position_d[3 * i] = pos[0];
        position_d[3 * i + 1] = pos[1];
        position_d[3 * i + 2] = pos[2];

        // Set the forces for each particle to 0
        force_d[3 * i] = 0.0;
        force_d[3 * i + 1] = 0.0;
        force_d[3 * i + 2] = 0.0;

        // Get particle type.
        type_d[i] = particles[i].getType();
    }
    
    // Launch the kernel with one thread per particle.
    int threadsPerBlock = 256;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    computeLJForces<<<numBlocks, threadsPerBlock>>>(position_d, force_d, type_d, n);
    hipDeviceSynchronize();
    MolecularDynamics::setParticleForces();
}

void MolecularDynamics::forwardEuler() {
    for (Particle& p : particles) {
        std::array<double, 3> velocity = p.getVelocity();
        std::array<double, 3> position = p.getPosition();
        const std::array<double, 3> force = p.getForce();
        const double mass = p.getMass();
        for (int k = 0; k < 3; ++k) {
            velocity[k] += dt * force[k] / mass;
            position[k] += dt * velocity[k];
        }
        p.setVelocity(velocity);
        p.setPosition(position);
    }
    bcCheck();
}

void MolecularDynamics::setParticleForces() {
    int n = particles.size();
    std::array<double, 3> particleForce;
    double fx, fy, fz;

    for (int i = 0; i < n; ++i) {
        fx = force_d[3 * i];
        fy = force_d[3 * i + 1];
        fz = force_d[3 * i + 2];
        particleForce = {fx, fy, fz};
        particles[i].setForce(particleForce);
    }
}

void MolecularDynamics::bcCheck() {
    // Reflect particles if they go out of bounds
    for (Particle& p : particles) {
        std::array<double, 3> position = p.getPosition();
        std::array<double, 3> velocity = p.getVelocity();
        // For each coordinate: 0->Lx, 1->Ly, 2->Lz
        for (int k = 0; k < 3; ++k) {
            double L = (k == 0) ? Lx : ((k == 1) ? Ly : Lz); // Select the correct bound size for each axis
            if (position[k] < 0) {
                position[k] = -position[k];
                velocity[k] = std::abs(velocity[k]);
            } else if (position[k] > L) {
                position[k] = 2 * L - position[k];
                velocity[k] = -std::abs(velocity[k]);
            }
        }
        p.setPosition(position);
        p.setVelocity(velocity);
    }
}

/**
 * @brief Calculates the total kinetic energy of the system.
 *
 * This function iterates over all particles in the system at current 
 * time, retrieves their velocities, and computes the kinetic energy 
 * using the formula:
 * 
 *     KE = 0.5 * mass * (velocity_x^2 + velocity_y^2 + velocity_z^2)
 * 
 * The total kinetic energy is then accumulated and stored in the member
 * variable `kineticEnergy`.
 */
void MolecularDynamics::calKE() {
    kineticEnergy = 0.0;
    for (const Particle& p : particles) {
        const std::array<double, 3>& velocity = p.getVelocity();
        double speedSquared = velocity[0] * velocity[0] + velocity[1] * velocity[1] + velocity[2] * velocity[2]; 
        kineticEnergy += 0.5 * p.getMass() * speedSquared;
    }
}

void MolecularDynamics::velRescale() {
    // check if temp is not set else continue
    if (temp == -1.0) {
        return;
    }

    constexpr double kb = 0.8314459920816467; // Boltzmann constant
    double tempKE = (2.0 / (3.0 * kb * N)) * kineticEnergy;
    const double lambda = std::sqrt(temp / tempKE);

    for (Particle& p : particles) {
        std::array<double, 3> velocity = p.getVelocity();
        for (int k = 0; k < 3; ++k) {
            velocity[k] *= lambda;
        }
        p.setVelocity(velocity);
    }
}

void MolecularDynamics::runSimulation() {
    // Compute initial kinetic energy
    calKE();
    
    // Output initial conditions before starting integration such that
    // the initial state is also recorded at time 0.
    outputParticleData(0);
    outputKineticEnergy(0);

    const double outputTime = 0.1; // output time for the txt files
    const int steps = static_cast<int>(finalTime / dt); // simulation time steps
    const int outputStep = static_cast<int>(outputTime / dt); // file output time steps

    // Start simulation from step 1
    for (int step = 1; step <= steps; ++step) {
        const double currentTime = step * dt;
        calKE();

        if (testCase == -1) 
            velRescale();
        
        calForcesCUDA();

        forwardEuler();

        // Output particle data and kinetic energy at 0.1 unit time intervals for the output files
        if (step % outputStep == 0) {
            if (testCase != -1) {
                outputParticleData(currentTime);
            }
            outputKineticEnergy(currentTime);
        }
    }
}

void MolecularDynamics::outputParticleData(double time) {
    for (size_t i = 0; i < particles.size(); ++i) {
        const Particle& p = particles[i];
        writeFile.writeParticleData(time, i, p.getPosition(), p.getVelocity());
    }
}

void MolecularDynamics::outputKineticEnergy(double time) {
    writeFile.writeKineticEnergy(time, kineticEnergy);
}
