#include "hip/hip_runtime.h"
/**
 * @file md.cu
 * @brief Implementation of the MolecularDynamics class.
 * 
 * This file contains the implementation of the MolecularDynamics class, which
 * is responsible for simulating molecular dynamics. The class includes methods
 * for initializing particles, computing forces, updating positions and velocities,
 * and handling boundary conditions.
 * 
 * The implementation leverages CUDA for parallel computation to efficiently
 * handle large numbers of particles and interactions.
 * 
 * @note Ensure that the CUDA runtime and necessary libraries are properly
 * installed and configured before compiling and running this code.
 */

#include "../include/md.h"
#include <ctime>
#include <cmath>
#include <omp.h>
#include <iostream>

// I acknowledge the use of ChatGPT 4.0 to generate an outline of how the structure of the MolecularDynamics class should be implemented.
// The majority of the content inside the methods was implemented by me. Where the code that was produced by AI is marked with a comment inside
// the relevant method, 

/**
 * @brief Compute Lennard-Jones forces for a set of particles.
 *
 * This kernel computes the Lennard-Jones forces acting on each particle
 * based on their positions and types. The forces are stored in the provided
 * force array.
 *
 * @param position Pointer to the array of particle positions. The array should
 *                 be of size 3 * n, where n is the number of particles.
 * @param force    Pointer to the array where the computed forces will be stored.
 *                 The array should be of size 3 * n.
 * @param type     Pointer to the array of particle types. The array should be
 *                 of size n.
 * @param n        The number of particles.
 */
__global__ void calLJForces(double* position, double* force, int* type, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    
    double force_accum_x = 0.0, force_accum_y = 0.0, force_accum_z = 0.0;
    
    // Cache particle i position and type to reduce repeated lookups
    double posx_i = position[3 * i];
    double posy_i = position[3 * i + 1];
    double posz_i = position[3 * i + 2];
    int type_i = type[i];
    
    // Loop over all particles to compute interactions.
    for (int j = 0; j < n; ++j) {
        if (i == j) continue;

        // Cache particle j position and type to reduce repeated lookups
        double posx_j = position[3 * j];
        double posy_j = position[3 * j + 1];
        double posz_j = position[3 * j + 2];
        int type_j = type[j];
        
        // Compute the vector between particles and squared distance
        double dx = posx_i - posx_j;
        double dy = posy_i - posy_j;
        double dz = posz_i - posz_j;
        double r2 = dx * dx + dy * dy + dz * dz;
        
        // Find correct parameters for sigma and epsilon based on particle types
        double sigma2, epsilon;
        if (type_i == 0 && type_j == 0) {
            sigma2 = 1.0;
            epsilon = 3.0;
        } else if (type_i == 1 && type_j == 1) {
            sigma2 = 9.0;
            epsilon = 60.0;
        } else {
            sigma2 = 4.0;
            epsilon = 15.0;
        }
        
        // Here AI was used to find the most efficient way to represent the variables that is to be
            // used in the force calculation, specifically the use of sigma6, sigma12, inv_r2, inv_r6, inv_r12
            // Pre-compute powers and inverses
        double sigma6 = sigma2 * sigma2 * sigma2;
        double sigma12 = sigma6 * sigma6;
        double inv_r2 = 1.0 / r2;
        double inv_r6 = inv_r2 * inv_r2 * inv_r2;
        double inv_r12 = inv_r6 * inv_r6;
        
        double f = 24.0 * epsilon * inv_r2 * (2.0 * sigma12 * inv_r12 - sigma6 * inv_r6);
        
        force_accum_x += f * dx;
        force_accum_y += f * dy;
        force_accum_z += f * dz;

        //printf("i,j %d,%d Force: (%f, %f, %f)\n", i, j, f * dx, dy, dz);
    }
    
    // Write the net force for particle i to global memory.
    force[3 * i]     = force_accum_x;
    force[3 * i + 1] = force_accum_y;
    force[3 * i + 2] = force_accum_z;
}


/**
 * @brief Constructs a MolecularDynamics object with the specified parameters.
 * 
 * @param numParticles The number of particles in the simulation.
 * @param dt The time step for the simulation.
 * @param Lx The length of the simulation box in the x-dimension.
 * @param Ly The length of the simulation box in the y-dimension.
 * @param Lz The length of the simulation box in the z-dimension.
 * @param testCase The test case identifier for initializing the simulation.
 * @param temp The initial temperature of the system.
 * @param percent_type1 The percentage of particles of type 1.
 * @param finalTime The final time for the simulation.
 */
MolecularDynamics::MolecularDynamics(int numParticles, double dt, double Lx, double Ly, double Lz, int testCase, 
    double temp, double percent_type1, double finalTime)
    : N(numParticles), dt(dt), Lx(Lx), Ly(Ly), Lz(Lz), testCase(testCase), temp(temp), percent_type1(percent_type1), finalTime(finalTime), 
    writeFile("particle_data.txt", "kinetic_energy.txt"), position_d(nullptr), force_d(nullptr), type_d(nullptr) {
    std::srand(static_cast<unsigned int>(std::time(nullptr))); 
    initialiseParticles();
}

/**
 * @brief Destructor for the MolecularDynamics class.
 *
 * This destructor is responsible for cleaning up any resources
 * allocated to the GPU. It ensures that
 * all memory are properly released when
 * an instance of the class is destroyed.
 */
MolecularDynamics::~MolecularDynamics() {
    // Free GPU memory
    hipFree(position_d);
    hipFree(force_d);
    hipFree(type_d);
}

/**
 * @brief Initialises the position, velocity and mass of particles for the molecular dynamics simulation based on the specified test case.
 * 
 * This function first clears any existing particles and initialises new particles based on the value of the `testCase` member variable.
 * For each test case, the memory for the position, force and type arrays are allocated on the GPU using `hipMallocManaged`.
 * 
 * Test cases:
 * - 1: Initialies a single particle at position (10.0, 10.0, 10.0) with zero velocity.
 * - 2: Initialises a single particle at position (10.0, 10.0, 10.0) with velocity (5.0, 2.0, 1.0).
 * - 3: Initialises two particles at positions (8.5, 10.0, 10.0) and (11.5, 10.0, 10.0) with zero velocity.
 * - 4: Initialises two particles at positions (8.5, 11.5, 10.0) and (11.5, 8.5, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively.
 * - 5: Initialises two particles at positions (8.5, 11.3, 10.0) and (11.5, 8.7, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively.
 * - 6: Initialises two particles at positions (8.5, 11.3, 10.0) and (11.5, 8.7, 10.0) with velocities (0.5, 0.0, 0.0) and (-0.5, 0.0, 0.0) respectively, and type 1 mass.
 * - Random: Initialises a specified number of particles of type 0 and type 1 with random positions and velocities, ensuring the positions between
 *  particles isn't <= 5 unit length. This is done via the `stabilityCheck` function.
 * 
 * The number of particles and its type is determined by the `percent_type1` and `N` member variables.
 * 
 * @note The random positions and velocities are generated using the `rand()` function and to ensure true randomness at every generation, a seed using the current time was used.
 */
// Here AI was used to generate the code to set the positions and velocity since 
// it is a repetitive task, but the structure of the conditional statement was implemented by me.
void MolecularDynamics::initialiseParticles() {
    particles.clear(); // Clear any existing particles

    if (testCase == 1) {
        particles = {
            Particle({10.0, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 2) {
        particles = {
            Particle({10.0, 10.0, 10.0}, {5.0, 2.0, 1.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 3) {
        particles = {
            Particle({8.5, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0),
            Particle({11.5, 10.0, 10.0}, {0.0, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 4) {
        particles = {
            Particle({8.5, 11.5, 10.0}, {0.5, 0.0, 0.0}, 0),
            Particle({11.5, 8.5, 10.0}, {-0.5, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 5) {
        particles = {
            Particle({8.5, 11.3, 10.0}, {0.5, 0.0, 0.0}, 0),
            Particle({11.5, 8.7, 10.0}, {-0.5, 0.0, 0.0}, 0)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else if (testCase == 6) {
        particles = {
            Particle({8.5, 11.3, 10.0}, {0.5, 0.0, 0.0}, 1),
            Particle({11.5, 8.7, 10.0}, {-0.5, 0.0, 0.0}, 1)
        };
        int N = particles.size();
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));
    } else {
        int numType1 = static_cast<int>(std::ceil(percent_type1 / 100.0 * N));
        int numType0 = N - numType1;
        // Allocate GPU memory on the device
        hipMallocManaged(&position_d, 3 * N * sizeof(double));
        hipMallocManaged(&force_d, 3 * N * sizeof(double));
        hipMallocManaged(&type_d, N * sizeof(int));

        assignRandomStates(numType1, 1);
        assignRandomStates(numType0, 0);
}
}

/**
 * @brief Assigns random positions and velocities to a specified number of particles of a given type.
 *
 * This function generates random positions and velocities for a specified number of particles
 * of a given type. The positions are generated within the bounds of the simulation box defined
 * by Lx, Ly, and Lz. The velocities are generated randomly with components in the range [-0.5, 0.5].
 * The function ensures that the generated positions are valid by checking them with the stabilityCheck function.
 *
 * @param numType The number of particles to assign random states to.
 * @param type The type of particles to assign random states to.
 */
void MolecularDynamics::assignRandomStates(const int numType, const int type) {
    const double invRand = 1.0 / RAND_MAX;
    for (int i = 0; i < numType; ++i) {
        bool validPosition = false;
        std::array<double, 3> position, velocity;

        while (!validPosition) {
            position = { Lx * ((double)rand() * invRand), Ly * ((double)rand() * invRand), Lz * ((double)rand() * invRand) };
            velocity = { ((double)rand() * invRand - 0.5), ((double)rand() * invRand - 0.5), ((double)rand() * invRand - 0.5) };

            validPosition = stabilityCheck(position);
        }

        particles.emplace_back(position, velocity, type);
    }
}

// Here AI was used to outline implementation of how to detect whether particles are too close 
// to each other, specifically the norm of the difference between the positions of two particles.
/**
 * @brief Checks the stability of a given position within the molecular dynamics system.
 *
 * This function determines whether a given position is stable by ensuring that it is not
 * too close to any existing particles in the system. The minimum allowed distance between
 * particles is defined by the constant R2 (0.25, which is 0.5 squared).
 *
 * @param position The position to check, represented as an array of three doubles.
 * @return true if the position is stable (i.e., not too close to any other particle), false otherwise.
 */
bool MolecularDynamics::stabilityCheck(const std::array<double, 3>& position) {
    constexpr double R2 = 0.25; // Minimum allowed distance squared (0.5^2) {
    for (const Particle& p : particles) {
        const std::array<double, 3> otherPosition = p.getPosition();
        double dx = position[0] - otherPosition[0];
        double dy = position[1] - otherPosition[1];
        double dz = position[2] - otherPosition[2];
        if ((dx*dx + dy*dy + dz*dz) < R2) {
            return false;
        }
    }
    return true;
}


/**
 * @brief Calculates the forces acting on particles in the molecular dynamics simulation using CUDA.
 * 
 * This function utilizes CUDA to perform parallel computations of the forces between particles.
 * First, the function copies the current particle positions and types into unified memory arrays,
 * resets the force array to zero, and then launches the CUDA kernel "calLJForces". In this kernel,
 * each thread is assigned to a single particle and computes its net force by iterating over all
 * other particles using the Lennard-Jones potential. Predefined lookup tables are used for the
 * interaction parameters. After synchronizing the device, the computed forces are copied back to
 * the particle data structure via setForce().
 */
void MolecularDynamics::calForcesCUDA() {
    const int n = particles.size();
    
    // Populate managed arrays from particle data.
    for (int i = 0; i < n; i++) {
        const std::array<double, 3>& pos = particles[i].getPosition();

        // Get the position of the particle
        position_d[3 * i] = pos[0];
        position_d[3 * i + 1] = pos[1];
        position_d[3 * i + 2] = pos[2];

        // Set the forces for each particle to 0
        force_d[3 * i] = 0.0;
        force_d[3 * i + 1] = 0.0;
        force_d[3 * i + 2] = 0.0;

        // Get particle type.
        type_d[i] = particles[i].getType();
    }
    
    // Launch the kernel with one thread per particle.
    int threadsPerBlock = 256;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    calLJForces<<<numBlocks, threadsPerBlock>>>(position_d, force_d, type_d, n);
    hipDeviceSynchronize();

    for (int i = 0; i < n; ++i) {
        std::array<double, 3> accum_force = {0.0,0.0,0.0};
        accum_force[0] = force_d[3 * i];
        accum_force[1] = force_d[3 * i + 1];
        accum_force[2] = force_d[3 * i + 2];
        particles[i].setForce(accum_force);
    }    
}

/**
 * @brief Advances the state of the molecular dynamics simulation using the forward Euler method.
 *
 * This method updates the velocity and position of each particle in the simulation based on the
 * current forces acting on them. The forward Euler method is a simple numerical integration technique
 * that approximates the new state of the system over a small time step (dt).
 *
 * The velocity of each particle is updated using the formula:
 *     v_new = v_old + (dt * force / mass)
 *
 * The position of each particle is then updated using the formula:
 *     x_new = x_old + (dt * v_new)
 *
 * After updating the velocities and positions, the method checks for boundary conditions.
 *
 * @note This method assumes that the particles have already been initialized with their respective
 *       velocities, positions, forces, and masses.
 */
void MolecularDynamics::forwardEuler() {
    for (Particle& p : particles) {
        std::array<double, 3> velocity = p.getVelocity();
        std::array<double, 3> position = p.getPosition();
        const std::array<double, 3> force = p.getForce();
        const double mass = p.getMass();
        for (int k = 0; k < 3; ++k) {
            velocity[k] += dt * force[k] / mass;
            position[k] += dt * velocity[k];
        }
        p.setVelocity(velocity);
        p.setPosition(position);
    }
    bcCheck();
}

/**
 * @brief Checks and applies boundary conditions to particles in the simulation.
 *
 * This function iterates over all particles in the simulation and ensures that
 * they remain within the defined simulation box boundaries. If a particle goes
 * out of bounds, it is reflected back into the simulation box, and its velocity
 * is adjusted accordingly to simulate a reflective boundary condition.
 *
 * The simulation box is defined by the dimensions Lx, Ly, and Lz along the x, y,
 * and z axes, respectively. For each particle, the function checks its position
 * along each axis:
 * - If the position is less than 0, the particle is reflected back into the box
 *   by setting its position to the negative of its current position and its
 *   velocity to the absolute value of its current velocity.
 * - If the position is greater than the box dimension, the particle is reflected
 *   back into the box by setting its position to twice the box dimension minus
 *   its current position and its velocity to the negative absolute value of its
 *   current velocity.
 *
 * After adjusting the position and velocity of a particle, the function updates
 * the particle's state.
 */
void MolecularDynamics::bcCheck() {
    // Reflect particles if they go out of bounds
    for (Particle& p : particles) {
        std::array<double, 3> position = p.getPosition();
        std::array<double, 3> velocity = p.getVelocity();
        // For each coordinate: 0->Lx, 1->Ly, 2->Lz
        for (int k = 0; k < 3; ++k) {
            // Here AI was used only to help define L in a very compact and efficient form.
            double L = (k == 0) ? Lx : ((k == 1) ? Ly : Lz); // Select the correct bound size for each axis
            if (position[k] < 0) {
                position[k] = -position[k];
                velocity[k] = std::abs(velocity[k]);
            } else if (position[k] > L) {
                position[k] = 2 * L - position[k];
                velocity[k] = -std::abs(velocity[k]);
            }
        }
        p.setPosition(position);
        p.setVelocity(velocity);
    }
}

/**
 * @brief Calculates the total kinetic energy of the system.
 *
 * This function iterates over all particles in the system at current 
 * time, retrieves their velocities, and computes the kinetic energy 
 * using the formula:
 * 
 *     KE = 0.5 * mass * (velocity_x^2 + velocity_y^2 + velocity_z^2)
 * 
 * The total kinetic energy is then accumulated and stored in the member
 * variable `kineticEnergy`.
 */
void MolecularDynamics::calKE() {
    kineticEnergy = 0.0;
    for (const Particle& p : particles) {
        const std::array<double, 3>& velocity = p.getVelocity();
        double speedSquared = velocity[0] * velocity[0] + velocity[1] * velocity[1] + velocity[2] * velocity[2]; 
        kineticEnergy += 0.5 * p.getMass() * speedSquared;
    }
}

/**
 * @brief Rescales the velocities of particles to match the desired temperature.
 *
 * This function rescales the velocities of all particles in the system to ensure
 * that the kinetic temperature matches the desired temperature (`temp`). If the
 * temperature (`temp`) is not set (i.e., it is -1.0), the function returns immediately.
 *
 * The rescaling factor (`lambda`) is calculated based on the ratio of the desired
 * temperature to the current kinetic temperature. Each component of the velocity
 * of every particle is then multiplied by this factor.
 *
 * @note The Boltzmann constant (`kb`) is defined as 0.8314459920816467.
 */
void MolecularDynamics::velRescale() {
    // check if temp is not set else continue
    if (temp == -1.0) {
        return;
    }

    constexpr double kb = 0.8314459920816467; // Boltzmann constant
    double tempKE = (2.0 / (3.0 * kb * N)) * kineticEnergy;
    const double lambda = std::sqrt(temp / tempKE);

    for (Particle& p : particles) {
        std::array<double, 3> velocity = p.getVelocity();
        for (int k = 0; k < 3; ++k) {
            velocity[k] *= lambda;
        }
        p.setVelocity(velocity);
    }
}

/**
 * @brief Runs the molecular dynamics simulation.
 * 
 * This function initializes the kinetic energy and outputs the initial conditions.
 * It then iterates over the simulation time steps, updating the system state using
 * the forward Euler method and computing forces using CUDA. The particle data and 
 * kinetic energy are output at specified time intervals.
 * 
 * @details
 * - Computes initial kinetic energy and outputs initial conditions.
 * - Iterates over the simulation time steps:
 *   - Updates the current time.
 *   - Computes kinetic energy.
 *   - Computes forces using CUDA.
 *   - Updates particle positions and velocities using forward Euler method.
 *   - Outputs particle data and kinetic energy at specified intervals.
 * 
 * @note If `testCase` is -1, velocity rescaling is performed.
 * 
 * @pre The simulation parameters such as `finalTime`, `dt`, and `testCase` must be set.
 * @pre The output functions `outputParticleData` and `outputKineticEnergy` must be defined.
 * 
 * @param None
 * @return None
 */
void MolecularDynamics::runSimulation() {
    // Compute initial kinetic energy
    calKE();
    
    // Output initial conditions before starting integration such that
    // the initial state is also recorded at time 0.
    if (testCase != -1) {
        outputParticleData(0);
    }
    outputKineticEnergy(0);

    const double outputTime = 0.1; // output time for the txt files
    const int steps = static_cast<int>(finalTime / dt); // simulation time steps
    const int outputStep = static_cast<int>(outputTime / dt); // file output time steps

    // Start simulation from step 1
    for (int step = 1; step <= steps; ++step) {
        const double currentTime = step * dt;
        calKE();

        if (testCase == -1) {
            velRescale();
        }
        
        calForcesCUDA();

        forwardEuler();

        // Output particle data and kinetic energy at 0.1 unit time intervals for the output files
        if (step % outputStep == 0) {
            if (testCase != -1) {
                outputParticleData(currentTime);
            }
            outputKineticEnergy(currentTime);
        }
    }
}

/**
 * @brief Outputs the data of all particles at a given time.
 *
 * This function iterates over all particles in the system and writes their
 * data (position and velocity) to a file.
 *
 * @param time The current simulation time.
 */
void MolecularDynamics::outputParticleData(double time) {
    for (int i = 0; i < particles.size(); ++i) {
        const Particle& p = particles[i];
        writeFile.writeParticleData(time, i, p.getPosition(), p.getVelocity());
    }
}


/**
 * @brief Outputs the kinetic energy of the system at a given time.
 * 
 * This function writes the current kinetic energy of the molecular dynamics
 * system to a file, associating it with the specified time.
 * 
 * @param time The current time at which the kinetic energy is being recorded.
 */
void MolecularDynamics::outputKineticEnergy(double time) {
    writeFile.writeKineticEnergy(time, kineticEnergy);
}
