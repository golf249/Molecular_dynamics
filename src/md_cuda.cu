#include "hip/hip_runtime.h"
#include "../include/md.h"

__global__ void calculateLJForces(double* position, double* force, double* type, int n) {
    // Get the index of the particle
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        for (int j = i+1; j < n; ++j) {
            
        }
    }
}

void MolecularDynamics::calForcesCUDA() {
    const int n = particles.size();

    // Allocate GPU memory on the device
    double* position;
    double* force;
    double* type;
    hipMallocManaged(&position, 3 * n * sizeof(double));
    hipMallocManaged(&force, 3 * n * sizeof(double));
    hipMallocManaged(&type, n * sizeof(double));

    // Populate the arrays in the allocated memory
    for (int i = 0; i < n; i++) {
        const std::array<double, 3>& pos = particles[i].getPosition();
        // Get the position of the particle
        position[3 * i] = pos[0];
        position[3 * i + 1] = pos[1];
        position[3 * i + 2] = pos[2];
        // Set the forces for each particle to 0
        force[3 * i] = 0.0;
        force[3 * i + 1] = 0.0;
        force[3 * i + 2] = 0.0;
        // Get the type of the particle
        type[i] = particles[i].getType();
    }

    // Calculate num thread and thread blocks
    int threads = std::min(256, n);
    int blocks = std::max(n/256, 1);

    // Launch the kernal to compute the Leonard-Jones forces
    calculateLJForces<<<blocks, threads>>>(position, force, type, n);
}